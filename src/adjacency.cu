/*
 ============================================================================
 Name        : adjacency.cu
 Author      : Jesse Harder
 Supervisor	 : Dr. Christopher Henry, P. Eng.
 Date		 : August 23, 2016
 Version     : V 1.0
 Description : This program will create an adjacency matrix
 	 	 	 	 for two provided files of vectors
 License	 : Licensed under the Non-Profit Open Software License version 3.0
 1) Grant of Copyright License. Licensor grants You a worldwide, royalty-free,
 non-exclusive, sublicensable license, for the duration of the copyright, to do the following:

 a) to reproduce the Original Work in copies, either alone or as part of a collective work;

 b) to translate, adapt, alter, transform, modify, or arrange the Original Work, thereby
 creating derivative works ("Derivative Works") based upon the Original Work;

 c) to distribute or communicate copies of the Original Work and Derivative Works
 to the public, with the proviso that copies of Original Work or Derivative Works
 that You distribute or communicate shall be licensed under this Non-Profit Open Software
 License or as provided in section 17(d);

 d) to perform the Original Work publicly; and

 e) to display the Original Work publicly.

 2) Grant of Patent License. Licensor grants You a worldwide, royalty-free,
 non-exclusive, sublicensable license, under patent claims owned or controlled by
  the Licensor that are embodied in the Original Work as furnished by the Licensor,
  for the duration of the patents, to make, use, sell, offer for sale, have made,
   and import the Original Work and Derivative Works.

 3) Grant of Source Code License. The term "Source Code" means the preferred
 form of the Original Work for making modifications to it and all available
 documentation describing how to modify the Original Work. Licensor agrees to
 provide a machine-readable copy of the Source Code of the Original Work along
 with each copy of the Original Work that Licensor distributes. Licensor reserves
 the right to satisfy this obligation by placing a machine-readable copy of the
 Source Code in an information repository reasonably calculated to permit
 inexpensive and convenient access by You for as long as Licensor continues
 to distribute the Original Work.

 4) Exclusions From License Grant. Neither the names of Licensor, nor the names
 of any contributors to the Original Work, nor any of their trademarks or service
 marks, may be used to endorse or promote products derived from this Original Work
 without express prior permission of the Licensor. Except as expressly stated
  herein, nothing in this License grants any license to Licensor's trademarks,
  copyrights, patents, trade secrets or any other intellectual property. No patent
  license is granted to make, use, sell, offer for sale, have made, or import embodiments
  of any patent claims other than the licensed claims defined in Section 2. No license
 is granted to the trademarks of Licensor even if such marks are included in the Original
  Work. Nothing in this License shall be interpreted to prohibit Licensor from licensing
  under terms different from this License any Original Work that Licensor otherwise would
  have a right to license.

 5) External Deployment. The term "External Deployment" means the use, distribution, or
 communication of the Original Work or Derivative Works in any way such that the Original
 Work or Derivative Works may be used by anyone other than You, whether those works are
 distributed or communicated to those persons or made available as an application intended
 for use over a network. As an express condition for the grants of license hereunder,
 You must treat any External Deployment by You of the Original Work or a Derivative
 Work as a distribution under section 1(c).

 6) Attribution Rights. You must retain, in the Source Code of any Derivative Works
 that You create, all copyright, patent, or trademark notices from the Source Code of
 the Original Work, as well as any notices of licensing and any descriptive text
 identified therein as an "Attribution Notice." You must cause the Source Code for
 any Derivative Works that You create to carry a prominent Attribution Notice reasonably
 calculated to inform recipients that You have modified the Original Work.

 7) Warranty of Provenance and Disclaimer of Warranty. The Original Work is provided
 under this License on an "AS IS" BASIS and WITHOUT WARRANTY, either express or implied,
 including, without limitation, the warranties of non-infringement, merchantability or
 fitness for a particular purpose. THE ENTIRE RISK AS TO THE QUALITY OF THE ORIGINAL WORK
 IS WITH YOU. This DISCLAIMER OF WARRANTY constitutes an essential part of this License.
 No license to the Original Work is granted by this License except under this disclaimer.

 8) Limitation of Liability. Under no circumstances and under no legal theory, whether
 in tort (including negligence), contract, or otherwise, shall the Licensor be liable
 to anyone for any direct, indirect, special, incidental, or consequential damages of
 any character arising as a result of this License or the use of the Original Work
 including, without limitation, damages for loss of goodwill, work stoppage, computer
 failure or malfunction, or any and all other commercial damages or losses. This limitation
 of liability shall not apply to the extent applicable law prohibits such limitation.

 9) Acceptance and Termination. If, at any time, You expressly assented to this License,
 that assent indicates your clear and irrevocable acceptance of this License and all of
 its terms and conditions. If You distribute or communicate copies of the Original Work
 or a Derivative Work, You must make a reasonable effort under the circumstances to obtain
 the express assent of recipients to the terms of this License. This License conditions
 your rights to undertake the activities listed in Section 1, including your right to create
 Derivative Works based upon the Original Work, and doing so without honoring these terms and
 conditions is prohibited by copyright law and international treaty. Nothing in this License
 is intended to affect copyright exceptions and limitations (including "fair use" or "fair
 dealing"). This License shall terminate immediately and You may no longer exercise any of
 the rights granted to You by this License upon your failure to honor the conditions in Section 1(c).

 10) Termination for Patent Action. This License shall terminate automatically and You
 may no longer exercise any of the rights granted to You by this License as of the date
 You commence an action, including a cross-claim or counterclaim, against Licensor or any
 licensee alleging that the Original Work infringes a patent. This termination provision
 shall not apply for an action alleging patent infringement by combinations of the Original
  Work with other software or hardware.

 11) Jurisdiction, Venue and Governing Law. Any action or suit relating to this License
 may be brought only in the courts of a jurisdiction wherein the Licensor resides or in
 which Licensor conducts its primary business, and under the laws of that jurisdiction
 excluding its conflict-of-law provisions. The application of the United Nations Convention
 on Contracts for the International Sale of Goods is expressly excluded. Any use of the Original
 Work outside the scope of this License or after its termination shall be subject to the
 requirements and penalties of copyright or patent law in the appropriate jurisdiction.
 This section shall survive the termination of this License.

 12) Attorneys' Fees. In any action to enforce the terms of this License or seeking
 damages relating thereto, the prevailing party shall be entitled to recover its costs and
 expenses, including, without limitation, reasonable attorneys' fees and costs incurred in
 connection with such action, including any appeal of such action. This section shall survive
 the termination of this License.

 13) Miscellaneous. If any provision of this License is held to be unenforceable, such provision
 shall be reformed only to the extent necessary to make it enforceable.

 14) Definition of "You" in This License. "You" throughout this License, whether in upper or
 lower case, means an individual or a legal entity exercising rights under, and complying with
 all of the terms of, this License. For legal entities, "You" includes any entity that controls,
 is controlled by, or is under common control with you. For purposes of this definition, "control"
 means (i) the power, direct or indirect, to cause the direction or management of such entity,
 whether by contract or otherwise, or (ii) ownership of fifty percent (50%) or more of the outstanding
 shares, or (iii) beneficial ownership of such entity.

 15) Right to Use. You may use the Original Work in all ways not otherwise restricted or conditioned
  by this License or by law, and Licensor promises not to interfere with or be responsible for such uses by You.

 16) Modification of This License. This License is Copyright © 2005 Lawrence Rosen.
 Permission is granted to copy, distribute, or communicate this License without modification.
 Nothing in this License permits You to modify this License as applied to the Original Work or to
 Derivative Works. However, You may modify the text of this License and copy, distribute or communicate
 your modified version (the "Modified License") and apply it to other original works of authorship
 subject to the following conditions: (i) You may not indicate in any way that your Modified License
 is the "Open Software License" or "OSL" and you may not use those names in the name of your Modified
 License; (ii) You must replace the notice specified in the first paragraph above with the notice
 "Licensed under <insert your license name here>" or with a notice of your own that is not confusingly
 similar to the notice in this License; and (iii) You may not claim that your original works are open
 source software unless your Modified License has been approved by Open Source Initiative (OSI) and
 You comply with its license review and certification process.

 17) Non-Profit Amendment. The name of this amended version of the Open Software License ("OSL 3.0")
 is "Non-Profit Open Software License 3.0". The original OSL 3.0 license has been amended as follows:

 (a) Licensor represents and declares that it is a not-for-profit organization that derives no revenue
 whatsoever from the distribution of the Original Work or Derivative Works thereof, or from support
 or services relating thereto.

 (b) The first sentence of Section 7 ["Warranty of Provenance"] of OSL 3.0 has been stricken. For
 Original Works licensed under this Non-Profit OSL 3.0, LICENSOR OFFERS NO WARRANTIES WHATSOEVER.

 (c) In the first sentence of Section 8 ["Limitation of Liability"] of this Non-Profit OSL 3.0,
 the list of damages for which LIABILITY IS LIMITED now includes "direct" damages.

 (d) The proviso in Section 1(c) of this License now refers to this "Non-Profit Open Software
 License" rather than the "Open Software License". You may distribute or communicate the Original
 Work or Derivative Works thereof under this Non-Profit OSL 3.0 license only if You make the
 representation and declaration in paragraph (a) of this Section 17. Otherwise, You shall distribute or
 communicate the Original Work or Derivative Works thereof only under the OSL 3.0 license and You shall
 publish clear licensing notices so stating. Also by way of clarification, this License does not authorize
 You to distribute or communicate works under this Non-Profit OSL 3.0 if You received them under
 the original OSL 3.0 license.

 (e) Original Works licensed under this license shall reference "Non-Profit OSL 3.0"
 in licensing notices to distinguish them from works licensed under the original OSL 3.0 license.
 ============================================================================
 */


#include <hip/hip_runtime.h>
#include <iostream>
#include <stdlib.h>
#include <vector>		//Provides access to vector object, for flexibly sized arrays
#include <fstream>		//Read input and write output files
#include <string>		//Provides string object
#include <sstream>		//Provides methods for working with strings

float EPSILON = 0.4;

//Write the boolean adjacency matrix to an output file, adjMatrix.txt
void writeToFile(bool* adjMatrix, unsigned totalVectors){

	std::cout << "Writing to adjMatrix.txt ... ";
	//Write output of final intersection to file
	std::ofstream out("adjMatrix.txt");

	unsigned totalValues = totalVectors * totalVectors;

	//Each output row represents one vector's adjacency values
	//ie: all 1's in row 0 represent sets adjacent to set 0
	for(unsigned i = 0; i < totalValues; ++i){
		if(i > 0 && i % totalVectors == 0)
			out << std::endl;
		out << adjMatrix[i];
	}

	std::cout << "Finished." << std::endl;
}

int main(int argc, const char ** argv) {
	unsigned setSize = 0;
	unsigned vectorSize = 0;
	unsigned setCount = 0;

	//This section of code deals with input parameters from the command line


	//Store list of input files
	std::vector<std::string> fileName;

	//Set option values for each parameter entered
	for (unsigned i = 0; i < argc; ++i) {
		if (argv[i] == std::string("-o")) {
			//Declare the size of the feature vectors
			std::stringstream convert(argv[i + 1]);
			convert >> vectorSize;
			i++;

		}else if (argv[i] == std::string("-e")) {
			std::stringstream convert(argv[i + 1]);
			convert >> EPSILON;
			i++;

		}else if (argv[i] == std::string("-f")) {
			//Push all files after -f into fileName vector
			for (unsigned j = i + 1; j < argc; ++j) {
				fileName.push_back(argv[j]);

			}
			break;
		} else if (i > 0) {
			std::cout << "Unknown parameter " << argv[i] << "." << std::endl;
		}
	}

	//Check for valid vector size
	if (vectorSize < 1) {
		std::cerr
				<< "The number of elements in each feature vector must be > 0. Set this with the -o parameter.\n";
		exit(1);
	}

	if(fileName.size() == 0){
		fileName.push_back("data/objectDescriptions0.txt");
		fileName.push_back("data/objectDescriptions1.txt");
	}


	//Read in the data

	//Total number of elements across all Fundamental Subsets, does not include the Count attribute
	unsigned totalSize = 0;
	setCount = fileName.size();
	std::fstream myfile(fileName[0].c_str(), std::ios_base::in);
	double fileElement;

	//Get size of file
	while (myfile >> fileElement) {
		totalSize++;
	}

	myfile.close();
	//Get total size of all files combined
	totalSize *= setCount;

	//Number of Feature Vectors in each Fundamental Subset
	setSize = (totalSize / vectorSize) / setCount;
	unsigned totalVectors = setSize * setCount;
	double *data = new double[setSize * vectorSize * setCount];

	//Fill in array (adjMatrix) with values from input files
	for (unsigned i = 0; i < setCount; ++i) {
		unsigned z = 0;
		unsigned setIndex = i * vectorSize * setSize;
		std::fstream inputFile(fileName[i].c_str(), std::ios_base::in);
		std::cout << "Reading from " << fileName[i].c_str() << "..." << std::endl;
		while (inputFile >> fileElement) {
			data[z + setIndex] = fileElement;
			z++;
		}
		z = 0;
	}

	//Square epsilon
	EPSILON = EPSILON * EPSILON;
	//Declare array to store output
	bool* adjMatrix = new bool[totalVectors*totalVectors];
	for(unsigned i = 0; i < totalVectors*totalVectors; ++i){
		adjMatrix[i] = 0;

	}

	std::cout << "Creating adjacency matrix..." << std::endl;
	//Compare each vector with each other vector, to see if euclidDistance is less than epsilon
	for(unsigned i = 0; i < totalVectors; ++i){
		for(unsigned j = 0; j < totalVectors; ++j){
			double euclidDistance = 0;
			//Adjacency matrix is mirrored along the diagonal, so [j][i] = [i][j]
			if(j < i){
				adjMatrix[i * totalVectors + j] = adjMatrix[j * totalVectors + i];
				continue;
			}else{
				for(unsigned k = 0; k < vectorSize; ++k){

					euclidDistance += powf(data[i * vectorSize + k] - data[j * vectorSize + k], 2);

				}
				if(euclidDistance <= EPSILON && i != j){
					adjMatrix[i * totalVectors + j] = true;
				}
			}
		}
	}

	//Write the matrix to a text file
	writeToFile(adjMatrix, totalVectors);

	/* Free memory */
	free(adjMatrix);
	free(data);

	std::cout << "Job's done." << std::endl;
	return 0;
}
